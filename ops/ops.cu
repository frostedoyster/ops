#include "hip/hip_runtime.h"
#include <iostream>
#include <torch/extension.h>
#include <torch/script.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace torch;

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
    CHECK_CUDA(x);     \
    CHECK_CONTIGUOUS(x)

#define FULL_MASK 0xffffffff

__host__ __device__ int find_integer_divisor(int x, int bdim)
{
    return (x + bdim - 1) / bdim;
}

template <typename scalar_t>
__global__ void forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> X,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Y,
    const torch::PackedTensorAccessor32<int32_t, 1, torch::RestrictPtrTraits> indices,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> output)
{
    extern __shared__ char buffer[];
    size_t offset = 0;
    scalar_t *smem = reinterpret_cast<scalar_t *>(buffer + offset);
    offset += blockDim.x * sizeof(scalar_t);

    int32_t edge_start = indices[blockIdx.x];
    int32_t edge_end = 0;

    if (blockIdx.x == indices.size(0) - 1)
    {
        edge_end = Y.size(0) - 1;
    }
    else
    {
        edge_end = indices[blockIdx.x + 1];
    }

    // clear out shared memory storage...
    for (int32_t m = threadIdx.y; m < Y.size(1); m += blockDim.y)
    {
        for (int32_t channel = threadIdx.x; channel < X.size(1); channel += blockDim.x)
        {
            smem[m * X.size(1) + channel] = 0.0;
        }
    }

    __syncthreads();

    for (int32_t i = edge_start; i < edge_end; i++)
    {
        for (int32_t m = threadIdx.y; m < Y.size(1); m += blockDim.y)
        {
            scalar_t y = Y[i][m];

            for (int32_t channel = threadIdx.x; channel < X.size(1); channel += blockDim.x)
            {
                scalar_t x = X[i][channel];

                smem[m * X.size(1) + channel] += x * y;
            }
        }
    }

    __syncthreads();

    for (int32_t m = threadIdx.y; m < Y.size(1); m += blockDim.y)
    {
        for (int32_t channel = threadIdx.x; channel < X.size(1); channel += blockDim.x)
        {
            output[blockIdx.x][m][channel] = smem[m * X.size(1) + channel];
        }
    }
}

torch::Tensor forward_gpu(torch::Tensor X,
                          torch::Tensor Y,
                          torch::Tensor neighbour_indices,
                          int32_t natoms,
                          int32_t nthreadx,
                          int32_t nthready,
                          int32_t nthreadz)
{

    torch::Tensor output = torch::empty({natoms, Y.size(1), X.size(1)},
                                        torch::TensorOptions()
                                            .dtype(X.dtype())
                                            .device(X.device()));

    dim3 block_dim(natoms);

    dim3 grid_dim(nthreadx, nthready, 1);

    AT_DISPATCH_FLOATING_TYPES(
        X.type(), "forward_gpu", ([&]
                                  {
                    size_t total_buff_size = 0;

                    total_buff_size += X.size(1) * Y.size(1) * sizeof(scalar_t);

                    forward_kernel<scalar_t><<<block_dim, grid_dim, total_buff_size>>>(
                        X.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                        Y.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                        neighbour_indices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
                        output.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>()); }));

    hipDeviceSynchronize();

    return output;
}

#define NEIGHBOUR_NEDGES_PER_BLOCK 512

/*
This function takes a sorted input sender_list, which maps each edge to a node by index, and outputs the "boundaries" when the index pattern changes

This is required by the CUDA code so we can send all calculations per-node to a single block.

the function loads 1024 + 1 elements into shared memory, and then loops through the buffer twice. Once for even boundaries, once for odd boundaries.
*/

__global__ void calculate_neighbours_kernel(const torch::PackedTensorAccessor32<int32_t, 1, torch::RestrictPtrTraits> sender_list,
                                            torch::PackedTensorAccessor32<int32_t, 1, torch::RestrictPtrTraits> edge_indices)
{
    extern __shared__ char buffer[];
    size_t offset = 0;
    int32_t *smem = reinterpret_cast<int32_t *>(buffer + offset);

    int32_t block_start = blockIdx.x * NEIGHBOUR_NEDGES_PER_BLOCK;

    int32_t nedges = sender_list.size(0);

    // load all elements of senderlist needed by block into shared memory
    for (int i = threadIdx.x; i < NEIGHBOUR_NEDGES_PER_BLOCK + 1; i += blockDim.x)
    {
        int32_t idx = block_start + i;

        if (idx < nedges)
        {
            smem[i] = sender_list[idx];
        }
    }

    __syncthreads();

    // deal with even boundaries
    for (int i = 2 * threadIdx.x; i < NEIGHBOUR_NEDGES_PER_BLOCK; i += 2 * blockDim.x)
    {
        int32_t idx = block_start + i;

        if (idx + 1 < nedges)
        {
            int32_t loc1 = smem[i];
            int32_t loc2 = smem[i + 1];

            if (loc1 != loc2)
            {
                edge_indices[loc2] = idx + 1;
            }
        }
    }

    // deal with odd boundaries
    for (int i = 2 * threadIdx.x + 1; i < NEIGHBOUR_NEDGES_PER_BLOCK + 1; i += 2 * blockDim.x)
    {
        int32_t idx = block_start + i;

        if (idx + 1 < nedges)
        {
            int32_t loc1 = smem[i];
            int32_t loc2 = smem[i + 1];

            if (loc1 != loc2)
            {
                edge_indices[loc2] = idx + 1;
            }
        }
    }

    // deal with 0th element specifically, so we dont need to use torch::zeros
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
        edge_indices[0] = 0;
    }
}

torch::Tensor calculate_neighbours_gpu(torch::Tensor sender_list, int32_t natoms, int32_t nthreadx)
{

    torch::Tensor output_indices = torch::empty(natoms,
                                                torch::TensorOptions()
                                                    .dtype(sender_list.dtype())
                                                    .device(sender_list.device()));

    int nbx = find_integer_divisor(sender_list.size(0), NEIGHBOUR_NEDGES_PER_BLOCK);

    dim3 block_dim(nbx);

    // printf("block dim: %d\n", nbx);

    dim3 grid_dim(nthreadx, 1, 1);

    size_t total_buff_size = 0;

    total_buff_size += (NEIGHBOUR_NEDGES_PER_BLOCK + 1) * sizeof(int32_t);

    calculate_neighbours_kernel<<<block_dim, grid_dim, total_buff_size>>>(

        sender_list.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>(),
        output_indices.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>());

    hipDeviceSynchronize();

    return output_indices;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("calculate_neighbours", &calculate_neighbours_gpu, "computes neighbourlist starts from sender list.");
    m.def("forward", &forward_gpu, "ops forward GPU.");
}